#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "Sudoku/Kernel.cuh"
#include <iostream>

__global__ void PredetermineTilesKernel(int subDim, int dimension, char *boardIn, char *boardOut)
{
    int tileId = threadIdx.x;
    int row = tileId / dimension;
    int col = tileId % dimension;
    int subGrid = GET_SUB_GRID(tileId, subDim);
    int rowOffset = subDim * (subGrid / subDim); // Subgrid tiles logic
    int colOffset = subDim * (subGrid % subDim);

    __shared__ bool modified[MAX_DIM * MAX_DIM];
    __shared__ bool madeChange;

    do
    {
        modified[threadIdx.x] = false;

        if (boardIn[tileId] != 0)
        {
            // Tile already has a set value, skip it
            boardOut[tileId] = boardIn[tileId];
        }
        else
        {
            bool usedNums[MAX_DIM];
            for (int iVal = 0; iVal < dimension; ++iVal)
            {
                // Every tile in column
                char tile = boardIn[iVal * dimension + col];
                if (tile > 0)
                {
                    usedNums[tile - 1] = true;
                }

                // Every tile in row
                tile = boardIn[row * dimension + iVal];
                if (tile > 0)
                {
                    usedNums[tile - 1] = true;
                }

                // Every tile in subgrid, convert iVal into 1D index of board
                tile = boardIn[(rowOffset + (iVal / subDim)) * dimension + colOffset + (iVal % subDim)];
                if (tile > 0)
                {
                    usedNums[tile - 1] = true;
                }
            }

            char candidate = 0;
            for (int i = 0; i < dimension; i++)
            {
                if (!usedNums[i])
                {
                    if (candidate != 0)
                    {
                        // More than one candidate value
                        candidate = 0;
                        break;
                    }
                    else
                    {
                        candidate = i + 1;
                    }
                }
            }

            boardOut[tileId] = candidate;
            modified[threadIdx.x] = candidate != 0;
        }

        __syncthreads();

        if (threadIdx.x == 0)
        {
            madeChange = false;
            for (int i = 0; i < dimension * dimension; ++i)
            {
                if (modified[i])
                {
                    madeChange = true;
                    break;
                }
            }
        }

        __syncthreads();

        // Running another iteration, copy current output for next input
        if (madeChange)
        {
            boardIn[tileId] = boardOut[tileId];
        }

        __syncthreads();

    } while (madeChange);
}

Board* PredetermineTiles(Board *boardIn)
{
    int dimension = boardIn->GetDimension();
    int subDim = sqrt(dimension);
    char *boardArrIn = boardIn->GetBoardPointer();

    char *dev_boardIn, *dev_boardOut;

    hipMalloc((void **)&dev_boardIn, dimension * dimension * sizeof(char));
    hipMalloc((void **)&dev_boardOut, dimension * dimension * sizeof(char));

    hipMemcpy(dev_boardIn, boardArrIn, dimension * dimension * sizeof(char), hipMemcpyHostToDevice);
    
    PredetermineTilesKernel<<<1, dimension * dimension>>>(subDim, dimension, dev_boardIn, dev_boardOut);

    char *boardArrOut = new char[dimension * dimension];
    hipMemcpy(boardArrOut, dev_boardOut, dimension * dimension * sizeof(char), hipMemcpyDeviceToHost);

    hipFree(dev_boardIn);
    hipFree(dev_boardOut);

    Board *out = new Board(dimension, boardArrOut);
    return out;
}



__global__ void RankFitnessKernel(int chromosomes, int dimension, char *flattenedPop, int *fitnessCount)
{
    int tileId = threadIdx.x;
    int chromOffset = blockIdx.x * blockDim.x;
    int row = tileId / dimension;
    int col = tileId % dimension;

    __shared__ int errors[MAX_DIM * MAX_DIM];
    errors[threadIdx.x] = 0;

    char currTile = flattenedPop[chromOffset + tileId];

    // Count how many tiles in col/row are same (conflicting) with current tile
    for (int iVal = 0; iVal < dimension; ++iVal)
    {
        int colTile = iVal * dimension + col;
        int rowTile = row * dimension + iVal;

        if (colTile != tileId && flattenedPop[chromOffset + colTile] == currTile)
        {
            ++errors[threadIdx.x];
        }

        if (rowTile != tileId && flattenedPop[chromOffset + rowTile] == currTile)
        {
            ++errors[threadIdx.x];
        }
    }

    __syncthreads();

    // Parallel reduction
    if (threadIdx.x % dimension == 0)
    {
        for (int i = 1; i < dimension; ++i)
        {
            errors[threadIdx.x] += errors[threadIdx.x + i];
        }
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        fitnessCount[blockIdx.x] = 0;
        for (int i = 0; i < dimension; ++i)
        {
            fitnessCount[blockIdx.x] += errors[i * dimension];
        }
    }

}

int* RankFitness(Population *popIn)
{
    int numChromosomes = 0;
    int numGenes = 0;

    // Arguments are output args, filled by function
    char *flattenedPop = popIn->FlattenPopulationToArray(numChromosomes, numGenes, false);

    int dimension = sqrt(numGenes);

    char *dev_flattenedPop;
    int *dev_fitnessCount;

    hipMalloc((void **)&dev_flattenedPop, numChromosomes * numGenes * sizeof(char));
    hipMalloc((void **)&dev_fitnessCount, numChromosomes * sizeof(int));

    hipMemcpy(dev_flattenedPop, flattenedPop, numChromosomes * numGenes * sizeof(char), hipMemcpyHostToDevice);

    RankFitnessKernel<<<numChromosomes, numGenes>>>(numChromosomes, dimension, dev_flattenedPop, dev_fitnessCount);

    int *fitnessRank = new int[numChromosomes];
    int *fitnessCount = new int[numChromosomes];

    hipMemcpy(fitnessCount, dev_fitnessCount, numChromosomes * sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(dev_flattenedPop);
    hipFree(dev_fitnessCount);

    // This can be parallelized but I was having some mem issues
    for (int i = 0; i < numChromosomes; ++i)
    {
        fitnessRank[i] = 0;
        for (int j = 0; j < numChromosomes; ++j)
        {   
            if (fitnessCount[i] > fitnessCount[j])
            {
                fitnessRank[i] += 1;
            }
        }
    }

    int* rank_set = new int[numChromosomes];
    // for (int i = 0; i < numChromosomes; ++i)
    // {
    //     rank_set[i] = -999;
    // }
    
    for (int i = 0; i < numChromosomes; ++i)
    {
        bool inSet = false;
        for (int j = 0; j < i; ++j)
        {
            if (rank_set[j] == fitnessRank[i])
            {
                inSet = true;
                break;
            }            
        }

        if (inSet)
        {
            fitnessRank[i] += 1;
        } 
        rank_set[i] = fitnessRank[i];
        std::cout << "(" << fitnessRank[i] << "\n";

    }
    return fitnessRank;
}


__global__ void OrderPopulation(int numChromosomes, int numGenes, char *flattenedPop, int *fitnessRank, int *gridswap, char *flattenedPopOut)
{
    
    int offset = ((blockIdx.x) * numGenes) + threadIdx.x;
    int rank_offset = (fitnessRank[blockIdx.x] * numGenes) + threadIdx.x;

    flattenedPopOut[rank_offset] = flattenedPop[offset];
} 

__global__ void BreedKernel(int numChromosomes, int numGenes, char *flattenedPop, int *fitnessRank, int *gridswap, char *flattenedPopOut)
{
    
    int swap = blockIdx.x % 2 ? -1 : 1;

    int grid = ((threadIdx.x % 9) / 3) + ((threadIdx.x / 27) * 3);
    int offset = ((blockIdx.x) * numGenes) + threadIdx.x;

    if (grid == gridswap[blockIdx.x])
    {
        int offset_new = ((blockIdx.x + swap) * numGenes) + threadIdx.x;
        flattenedPopOut[offset] = flattenedPop[offset_new];
    }
    else 
    {
        flattenedPopOut[offset] = flattenedPop[offset];
    }
}

Population* Breed(Population *popIn, int* rankings)
{
    double choices[2] = {0.20, 0.8}; // Keep top 20%, breed next 60%, drop final 20%

    int numChromosomes = 0;
    int numGenes = 0;
    std::shared_ptr<bool[]> lockedGenesIn = popIn->GetLockedGenes();

    // Arguments are output args, filled by function
    // Need to delete returned pointer at end
    char *flattenedPop = popIn->FlattenPopulationToArray(numChromosomes, numGenes);

    char *dev_flattenedPopBreed;
    char *dev_flattenedPopBreedOut;
    int *dev_fitnessRankBreed;
    int *dev_gridSwap;
    int *grid_swap = new int[numChromosomes];

    for (int i = 0; i < numChromosomes / 2; i++){
        int swap = rand() % 9;
        grid_swap[i * 2] = swap;
        grid_swap[(i * 2) + 1] = swap;

    }

    hipMalloc((void **)&dev_flattenedPopBreed, numChromosomes * numGenes * sizeof(char));
    hipMalloc((void **)&dev_flattenedPopBreedOut, numChromosomes * numGenes * sizeof(char));
    hipMalloc((void **)&dev_fitnessRankBreed, numChromosomes * sizeof(int));
    hipMalloc((void **)&dev_gridSwap, numChromosomes * sizeof(int));

    hipMemcpy(dev_flattenedPopBreed, flattenedPop, numChromosomes * numGenes * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_fitnessRankBreed, rankings, numChromosomes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_gridSwap, grid_swap, numChromosomes * sizeof(int), hipMemcpyHostToDevice);

    OrderPopulation<<<numChromosomes, numGenes>>>(numChromosomes, numGenes, dev_flattenedPopBreed, dev_fitnessRankBreed, dev_gridSwap, dev_flattenedPopBreedOut);
    // BreedKernel<<<numChromosomes, numGenes>>>(numChromosomes, numGenes, dev_flattenedPopBreed, dev_fitnessRankBreed, dev_gridSwap, dev_flattenedPopBreedOut);
    
    char *popout = new char[numChromosomes * numGenes];
    hipMemcpy(popout, dev_flattenedPopBreedOut, numChromosomes * numGenes * sizeof(char), hipMemcpyDeviceToHost);
    
    Population *out = new Population(numGenes, numChromosomes, lockedGenesIn, popout);
    // Population *out = new Population(numGenes, numChromosomes, lockedGenesIn, pop);
    return out;
}