#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "Sudoku/Kernel.cuh"
#include <iostream>
#include <algorithm>    // std::shuffle
#include <array>        // std::array
#include <random>       // std::default_random_engine
#include <chrono>       // std::chrono::system_clock
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void PredetermineTilesKernel(int subDim, int dimension, char *boardIn, char *boardOut)
{
    int tileId = threadIdx.x;
    int row = tileId / dimension;
    int col = tileId % dimension;
    int subGrid = GET_SUB_GRID(tileId, subDim);
    int rowOffset = subDim * (subGrid / subDim); // Subgrid tiles logic
    int colOffset = subDim * (subGrid % subDim);

    __shared__ bool modified[MAX_DIM * MAX_DIM];
    __shared__ bool madeChange;

    do
    {
        modified[threadIdx.x] = false;

        if (boardIn[tileId] != 0)
        {
            // Tile already has a set value, skip it
            boardOut[tileId] = boardIn[tileId];
        }
        else
        {
            bool usedNums[MAX_DIM];
            for (int iVal = 0; iVal < dimension; ++iVal)
            {
                // Every tile in column
                char tile = boardIn[iVal * dimension + col];
                if (tile > 0)
                {
                    usedNums[tile - 1] = true;
                }

                // Every tile in row
                tile = boardIn[row * dimension + iVal];
                if (tile > 0)
                {
                    usedNums[tile - 1] = true;
                }

                // Every tile in subgrid, convert iVal into 1D index of board
                tile = boardIn[(rowOffset + (iVal / subDim)) * dimension + colOffset + (iVal % subDim)];
                if (tile > 0)
                {
                    usedNums[tile - 1] = true;
                }
            }

            char candidate = 0;
            for (int i = 0; i < dimension; i++)
            {
                if (!usedNums[i])
                {
                    if (candidate != 0)
                    {
                        // More than one candidate value
                        candidate = 0;
                        break;
                    }
                    else
                    {
                        candidate = i + 1;
                    }
                }
            }

            boardOut[tileId] = candidate;
            modified[threadIdx.x] = candidate != 0;
        }

        __syncthreads();

        if (threadIdx.x == 0)
        {
            madeChange = false;
            for (int i = 0; i < dimension * dimension; ++i)
            {
                if (modified[i])
                {
                    madeChange = true;
                    break;
                }
            }
        }

        __syncthreads();

        // Running another iteration, copy current output for next input
        if (madeChange)
        {
            boardIn[tileId] = boardOut[tileId];
        }

        __syncthreads();

    } while (madeChange);
}

Board* PredetermineTiles(Board *boardIn)
{
    int dimension = boardIn->GetDimension();
    int subDim = sqrt(dimension);
    char *boardArrIn = boardIn->GetBoardPointer();

    char *dev_boardIn, *dev_boardOut;

    hipMalloc((void **)&dev_boardIn, dimension * dimension * sizeof(char));
    hipMalloc((void **)&dev_boardOut, dimension * dimension * sizeof(char));

    hipMemcpy(dev_boardIn, boardArrIn, dimension * dimension * sizeof(char), hipMemcpyHostToDevice);
    
    PredetermineTilesKernel<<<1, dimension * dimension>>>(subDim, dimension, dev_boardIn, dev_boardOut);

    char *boardArrOut = new char[dimension * dimension];
    hipMemcpy(boardArrOut, dev_boardOut, dimension * dimension * sizeof(char), hipMemcpyDeviceToHost);

    hipFree(dev_boardIn);
    hipFree(dev_boardOut);

    Board *out = new Board(dimension, boardArrOut);
    return out;
}



__global__ void RankFitnessKernel(int chromosomes, int dimension, char *flattenedPop, int *fitnessCount)
{
    int tileId = threadIdx.x;
    int chromOffset = blockIdx.x * blockDim.x;
    int row = tileId / dimension;
    int col = tileId % dimension;

    __shared__ int errors[MAX_DIM * MAX_DIM];
    errors[threadIdx.x] = 0;

    char currTile = flattenedPop[chromOffset + tileId];

    // Count how many tiles in col/row are same (conflicting) with current tile
    for (int iVal = 0; iVal < dimension; ++iVal)
    {
        int colTile = iVal * dimension + col;
        int rowTile = row * dimension + iVal;

        if (colTile != tileId && flattenedPop[chromOffset + colTile] == currTile)
        {
            ++errors[threadIdx.x];
        }

        if (rowTile != tileId && flattenedPop[chromOffset + rowTile] == currTile)
        {
            ++errors[threadIdx.x];
        }
    }

    __syncthreads();

    // Parallel reduction
    if (threadIdx.x % dimension == 0)
    {
        for (int i = 1; i < dimension; ++i)
        {
            errors[threadIdx.x] += errors[threadIdx.x + i];
        }
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        fitnessCount[blockIdx.x] = 0;
        for (int i = 0; i < dimension; ++i)
        {
            fitnessCount[blockIdx.x] += errors[i * dimension];
        }
    }

}

int* RankFitness(int numChromosomes, int numGenes, char *flattenedPop, int *errorCountsOut)
{
    // int numChromosomes = 0;
    // int numGenes = 0;

    // Arguments are output args, filled by function
    // char *flattenedPop = popIn->FlattenPopulationToArray(numChromosomes, numGenes, false);

    int dimension = sqrt(numGenes);

    char *dev_flattenedPop;
    int *dev_fitnessCount;

    hipMalloc((void **)&dev_flattenedPop, numChromosomes * numGenes * sizeof(char));
    hipMalloc((void **)&dev_fitnessCount, numChromosomes * sizeof(int));

    hipMemcpy(dev_flattenedPop, flattenedPop, numChromosomes * numGenes * sizeof(char), hipMemcpyHostToDevice);

    RankFitnessKernel<<<numChromosomes, numGenes>>>(numChromosomes, dimension, dev_flattenedPop, dev_fitnessCount);

    int *fitnessRank = new int[numChromosomes];

    hipMemcpy(errorCountsOut, dev_fitnessCount, numChromosomes * sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(dev_flattenedPop);
    hipFree(dev_fitnessCount);

    // This can be parallelized but I was having some mem issues
    for (int i = 0; i < numChromosomes; ++i)
    {
        fitnessRank[i] = 0;
        for (int j = 0; j < numChromosomes; ++j)
        {   
            if (errorCountsOut[i] > errorCountsOut[j])
            {
                fitnessRank[i] += 1;
            }
        }
    }

    int* rank_set = new int[numChromosomes];
    // for (int i = 0; i < numChromosomes; ++i)
    // {
    //     rank_set[i] = -999;
    // }
    
    for (int i = 0; i < numChromosomes; ++i)
    {
        while (true){

            bool inSet = false;

            for (int j = 0; j < i; ++j)
            {
                if (rank_set[j] == fitnessRank[i])
                {
                    inSet = true;
                    break;
                }            
            }

            if (inSet)
            {
                fitnessRank[i] += 1;
            } 
            else 
            {
                break;
            }
        }
        
        rank_set[i] = fitnessRank[i];
    }
    return fitnessRank;
}

__global__ void SelectionKernel(int numChromosomes, int numGenes, char *flattenedPop, int *ranks, char *flattenedPopOut)
{
    // int grid = ((threadIdx.x % 9) / 3) + ((threadIdx.x / 27) * 3); // Determines the grid of current run
    // if (ranks[blockIdx.x] <= (int)(numChromosomes * RANK_RETENTION_RATE))
    // {
        int offset = (blockIdx.x * numGenes) + threadIdx.x;
        int rank = (ranks[blockIdx.x] * numGenes) + threadIdx.x;
        flattenedPopOut[rank] = flattenedPop[offset];
    // }
}

__global__ void BreedKernel(int numChromosomes, int numGenes, int seed, char *flattenedPop, int *ranks, bool *lockedIn, int* dev_swap_index, int* dev_swap_candidates, char *flattenedPopOut)
{
    // Random mutations
    hiprandState_t state;
    
    if (threadIdx.x == 0) 
    {   
        /* we have to initialize the state */
        hiprand_init(0, /* the seed controls the sequence of random values that are produced */
                blockIdx.x, /* the sequence number is only important with multiple cores */
                0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                &state);

        int c1 = (hiprand(&state) % (int)(numChromosomes * RANK_RETENTION_RATE));
        int c2 = (hiprand(&state) % (int)(numChromosomes * RANK_RETENTION_RATE));
        int c3 = (hiprand(&state) % (int)(numChromosomes * RANK_RETENTION_RATE));
        int c4 = (hiprand(&state) % (int)(numChromosomes * RANK_RETENTION_RATE));


        dev_swap_index[blockIdx.x] = (blockIdx.x % 7) + 1;
        dev_swap_candidates[(blockIdx.x * 2)] = c1 < c2 ? c1 : c2;
        dev_swap_candidates[(blockIdx.x * 2) + 1] = c3 < c4 ? c3 : c4;
    }


    __syncthreads();

    int grid = ((threadIdx.x % 9) / 3) + ((threadIdx.x / 27) * 3); // Determines the grid of current run
    int offset = ((blockIdx.x) * numGenes) + threadIdx.x;
    
    if (grid < dev_swap_index[blockIdx.x])
    {
        int offset_new = (dev_swap_candidates[(blockIdx.x * 2)] * numGenes) + threadIdx.x;
        flattenedPopOut[offset] = flattenedPop[offset_new];
    }
    else 
    {
        int offset_new = (dev_swap_candidates[(blockIdx.x * 2) + 1]  * numGenes) + threadIdx.x;
        flattenedPopOut[offset] = flattenedPop[offset_new];
    }

    __syncthreads();

    if (threadIdx.x == 0) 
    {   
        for (int k = 0; k < (hiprand(&state) % 3) + 1; k++)
        {
            int swap_index_1 = (hiprand(&state) % 81);
         
            while(lockedIn[swap_index_1]) 
            {
                swap_index_1 = (hiprand(&state) % 81);
            }

            int swap_grid = ((swap_index_1 % 9) / 3) + ((swap_index_1 / 27) * 3); 
            int swap_grid_center = 10 + ((swap_grid / 3) * 27) + ((swap_grid % 3) * 3);

            int swap_index_2 = swap_index_1;

            do
            {
                int y_shift = (hiprand(&state) % 3) - 1;
                int x_shift = (hiprand(&state) % 3) - 1;

                swap_index_2 = swap_grid_center + (9 * y_shift) + x_shift;
            } while(swap_index_1 == swap_index_2 || lockedIn[swap_index_2]);
            
            char temp = flattenedPopOut[(blockIdx.x * 81) + swap_index_1];

            flattenedPopOut[(blockIdx.x * 81) + swap_index_1] = flattenedPopOut[(blockIdx.x * 81) + swap_index_2];
            flattenedPopOut[(blockIdx.x * 81) + swap_index_2] = temp;
        }
    }
}

Population* Breed(Population *popIn, int &bestrank, char* bestboard)
{
    double choices[2] = {0.20, 0.8}; // Keep top 20%, breed next 60%, drop final 20%

    int numChromosomes = 0;
    int numGenes = 0;
    std::shared_ptr<bool[]> lockedGenesIn = popIn->GetLockedGenes();

    // Arguments are output args, filled by function
    // Need to delete returned pointer at end
    char *flattenedPop = popIn->FlattenPopulationToArrayShuffle(numChromosomes, numGenes);

    int *errorCounts = new int[numChromosomes];
    int* fitnessRanks = RankFitness(numChromosomes, numGenes, flattenedPop, errorCounts);

    // for (int k = 0; k < numChromosomes; k++){
    //     std::cout << fitnessRanks[k] << ",";
    // }
    // std::cout << "\n";
    
    int prev_best = bestrank;
    bestrank = 999;
    int index = 0;
    for (int e = 0; e < numChromosomes; e++)
    {
        if (errorCounts[e] < bestrank) 
        {
            bestrank = errorCounts[e];
            index = e;
            if (bestrank < prev_best)
            {
                for (int i = 0; i < 9; i++)
                {
                    for (int j = 0; j < 9; j++)
                    {
                        bestboard[(i * 9) + j] = flattenedPop[(index * numGenes) + (i * 9) + j];
                    }
                }
            }
        }
        
    }

    std::cout << "Best error - " << bestrank << "\n";
    // for (int i = 0; i < 9; i++)
    // {
    //     for (int j = 0; j < 9; j++)
    //     {
    //         std::cout << (int)flattenedPop[(index * numGenes) + (i * 9) + j] << " ";
    //     }
    //     std::cout << "\n";
    // }
    // std::cout << "\n";
    
    char *dev_flattenedPopBreed;
    char *dev_tempPopualtion;
    char *dev_flattenedPopBreedOut;
    int *dev_ranks;
    bool *dev_lockedIn;
    int *dev_swap_index;
    int *dev_swap_candidates;

    int retention_size = (int)(numChromosomes * RANK_RETENTION_RATE);

    hipMalloc((void **)&dev_flattenedPopBreed, numChromosomes * numGenes * sizeof(char));
    hipMalloc((void **)&dev_flattenedPopBreedOut, numChromosomes * numGenes * sizeof(char));
    hipMalloc((void **)&dev_tempPopualtion, retention_size * numGenes * sizeof(char));
    hipMalloc((void **)&dev_ranks, numChromosomes * sizeof(int));
    // hipMalloc((void **)&dev_swaps, numChromosomes * sizeof(int));
    hipMalloc((void **)&dev_lockedIn, numChromosomes * sizeof(bool));
    hipMalloc((void **)&dev_swap_index, numChromosomes * sizeof(int));
    hipMalloc((void **)&dev_swap_candidates, numChromosomes * 2 * sizeof(int));

    hipMemcpy(dev_flattenedPopBreed, flattenedPop, numChromosomes * numGenes * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_ranks, fitnessRanks, numChromosomes * sizeof(int), hipMemcpyHostToDevice);

    SelectionKernel<<<numChromosomes, numGenes>>>(numChromosomes, numGenes, dev_flattenedPopBreed, dev_ranks, dev_tempPopualtion);

    // char* temp = new char[(int)(RANK_RETENTION_RATE * numChromosomes)];

    // hipMemcpy(temp, dev_tempPopualtion, (int)(RANK_RETENTION_RATE * numChromosomes * numGenes), hipMemcpyDeviceToHost);

    // for (int k = 0; k < (int)(RANK_RETENTION_RATE * numChromosomes); k++){
    //     std::cout << k << "\n";
    //     for (int i = 0; i < 9; i++)
    //     {
    //         for (int j = 0; j < 9; j++)
    //         {
    //             std::cout << (int)temp[(k * 81) + (i * 9) + j] << " ";
    //         }
    //         std::cout << "\n";
    //     }
    //     std::cout << "\n";
    // }
    
    hipMemcpy(dev_lockedIn, lockedGenesIn.get(), numChromosomes * sizeof(bool), hipMemcpyHostToDevice);

    // BreedKernel<<<numChromosomes, numGenes>>>(numChromosomes, numGenes, (rand() % 9) + 1, dev_flattenedPopBreed, dev_ranks, dev_lockedIn, dev_flattenedPopBreedOut);
    BreedKernel<<<numChromosomes, numGenes>>>(numChromosomes, numGenes, (rand() % 9) + 1, dev_tempPopualtion, dev_ranks, dev_lockedIn, dev_swap_index, dev_swap_candidates, dev_flattenedPopBreedOut);
    
    char *popout = new char[numChromosomes * numGenes];
    hipMemcpy(popout, dev_flattenedPopBreedOut, numChromosomes * numGenes * sizeof(char), hipMemcpyDeviceToHost);
    
    hipFree(dev_tempPopualtion);
    hipFree(dev_ranks);
    hipFree(dev_flattenedPopBreedOut);
    hipFree(dev_lockedIn);
    hipFree(dev_tempPopualtion);
    hipFree(dev_swap_index);
    hipFree(dev_swap_candidates);


    delete flattenedPop;
    delete errorCounts;
    delete fitnessRanks;

    Population *out = new Population(numGenes, numChromosomes, lockedGenesIn, popout);

    // delete popout;
    return out;
}