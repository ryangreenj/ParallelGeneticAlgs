#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "Sudoku/Kernel.cuh"
#include <iostream>
#include <algorithm>    // std::shuffle
#include <array>        // std::array
#include <random>       // std::default_random_engine
#include <chrono>       // std::chrono::system_clock
#include <map>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void PredetermineTilesKernel(int subDim, int dimension, char *boardIn, char *boardOut)
{
    int tileId = threadIdx.x;
    int row = tileId / dimension;
    int col = tileId % dimension;
    int subGrid = GET_SUB_GRID(tileId, subDim);
    int rowOffset = subDim * (subGrid / subDim); // Subgrid tiles logic
    int colOffset = subDim * (subGrid % subDim);

    __shared__ bool modified[MAX_DIM * MAX_DIM];
    __shared__ bool madeChange;

    do
    {
        modified[threadIdx.x] = false;

        if (boardIn[tileId] != 0)
        {
            // Tile already has a set value, skip it
            boardOut[tileId] = boardIn[tileId];
        }
        else
        {
            bool usedNums[MAX_DIM];
            for (int iVal = 0; iVal < dimension; ++iVal)
            {
                // Every tile in column
                char tile = boardIn[iVal * dimension + col];
                if (tile > 0)
                {
                    usedNums[tile - 1] = true;
                }

                // Every tile in row
                tile = boardIn[row * dimension + iVal];
                if (tile > 0)
                {
                    usedNums[tile - 1] = true;
                }

                // Every tile in subgrid, convert iVal into 1D index of board
                tile = boardIn[(rowOffset + (iVal / subDim)) * dimension + colOffset + (iVal % subDim)];
                if (tile > 0)
                {
                    usedNums[tile - 1] = true;
                }
            }

            char candidate = 0;
            for (int i = 0; i < dimension; i++)
            {
                if (!usedNums[i])
                {
                    if (candidate != 0)
                    {
                        // More than one candidate value
                        candidate = 0;
                        break;
                    }
                    else
                    {
                        candidate = i + 1;
                    }
                }
            }

            boardOut[tileId] = candidate;
            modified[threadIdx.x] = candidate != 0;
        }

        __syncthreads();

        if (threadIdx.x == 0)
        {
            madeChange = false;
            for (int i = 0; i < dimension * dimension; ++i)
            {
                if (modified[i])
                {
                    madeChange = true;
                    break;
                }
            }
        }

        __syncthreads();

        // Running another iteration, copy current output for next input
        if (madeChange)
        {
            boardIn[tileId] = boardOut[tileId];
        }

        __syncthreads();

    } while (madeChange);
}

Board* PredetermineTiles(Board *boardIn)
{
    int dimension = boardIn->GetDimension();
    int subDim = sqrt(dimension);
    char *boardArrIn = boardIn->GetBoardPointer();

    char *dev_boardIn, *dev_boardOut;

    hipMalloc((void **)&dev_boardIn, dimension * dimension * sizeof(char));
    hipMalloc((void **)&dev_boardOut, dimension * dimension * sizeof(char));

    hipMemcpy(dev_boardIn, boardArrIn, dimension * dimension * sizeof(char), hipMemcpyHostToDevice);
    
    PredetermineTilesKernel<<<1, dimension * dimension>>>(subDim, dimension, dev_boardIn, dev_boardOut);

    char *boardArrOut = new char[dimension * dimension];
    hipMemcpy(boardArrOut, dev_boardOut, dimension * dimension * sizeof(char), hipMemcpyDeviceToHost);

    hipFree(dev_boardIn);
    hipFree(dev_boardOut);

    Board *out = new Board(dimension, boardArrOut);
    return out;
}



__global__ void RankFitnessKernel(int chromosomes, int dimension, char *flattenedPop, int *fitnessCount)
{
    int tileId = threadIdx.x;
    int chromOffset = blockIdx.x * blockDim.x;
    int row = tileId / dimension;
    int col = tileId % dimension;

    __shared__ int errors[MAX_DIM * MAX_DIM];
    errors[threadIdx.x] = 0;

    char currTile = flattenedPop[chromOffset + tileId];

    // Count how many tiles in col/row are same (conflicting) with current tile
    for (int iVal = 0; iVal < dimension; ++iVal)
    {
        int colTile = iVal * dimension + col;
        int rowTile = row * dimension + iVal;

        if (colTile != tileId && flattenedPop[chromOffset + colTile] == currTile)
        {
            ++errors[threadIdx.x];
        }

        if (rowTile != tileId && flattenedPop[chromOffset + rowTile] == currTile)
        {
            ++errors[threadIdx.x];
        }
    }

    __syncthreads();

    // Parallel reduction
    if (threadIdx.x % dimension == 0)
    {
        for (int i = 1; i < dimension; ++i)
        {
            errors[threadIdx.x] += errors[threadIdx.x + i];
        }
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        fitnessCount[blockIdx.x] = 0;
        for (int i = 0; i < dimension; ++i)
        {
            fitnessCount[blockIdx.x] += errors[i * dimension];
        }
    }

}

int* RankFitness(int numChromosomes, int numGenes, char *flattenedPop, int *errorCountsOut)
{
    int dimension = sqrt(numGenes);

    char *dev_flattenedPop;
    int *dev_fitnessCount;

    hipMalloc((void **)&dev_flattenedPop, numChromosomes * numGenes * sizeof(char));
    hipMalloc((void **)&dev_fitnessCount, numChromosomes * sizeof(int));

    hipMemcpy(dev_flattenedPop, flattenedPop, numChromosomes * numGenes * sizeof(char), hipMemcpyHostToDevice);

    RankFitnessKernel<<<numChromosomes, numGenes>>>(numChromosomes, dimension, dev_flattenedPop, dev_fitnessCount);

    hipMemcpy(errorCountsOut, dev_fitnessCount, numChromosomes * sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(dev_flattenedPop);
    hipFree(dev_fitnessCount);

    // Give chromosomes a unique rank based on their errorCount/fitnessScore

    // Using std::sort is O(n log n), previous algorithm was O(n^2)
    // <fitnessScore, chromosomeIndex>
    std::vector<std::pair<int, int>> scoreToIndex = std::vector<std::pair<int, int>>();

    for (int i = 0; i < numChromosomes; ++i)
    {
        scoreToIndex.push_back(std::make_pair(errorCountsOut[i], i));
    }

    // Sorts by fitnessScore, O(log n)
    std::sort(scoreToIndex.begin(), scoreToIndex.end());

    int *fitnessRank = new int[numChromosomes];
    
    int i = 0;
    for (auto &pair : scoreToIndex)
    {
        fitnessRank[pair.second] = i++;
    }

    return fitnessRank;
}

__global__ void BreedKernel(int numChromosomes, int numGenes, int dimension, int subDim, int seed, char *flattenedPop, int *ranks, bool *lockedIn, int* dev_swap_index, int* dev_swap_candidates, char *dev_tempPopualtion)
{   

    hiprandState_t state;

    // Select top ranked solutions and place them in dev_tempPopualtion
    if (ranks[blockIdx.x] < (int)(numChromosomes * RANK_RETENTION_RATE))
    {
        int offset = (blockIdx.x * numGenes) + threadIdx.x;
        int rank = (ranks[blockIdx.x] * numGenes) + threadIdx.x;
        dev_tempPopualtion[rank] = flattenedPop[offset];
    }

    __syncthreads();
    // This block will pick the chromosomes that will be swaping, via a tournament style selection
    // The two solutions with the best rank will swap genes

    // This could easily be made to use an array and feature more than 4 prospective chomosomes

    if (threadIdx.x == 0) 
    {   
        /* we have to initialize the state */
        hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
                blockIdx.x, /* the sequence number is only important with multiple cores */
                0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                &state);

        // Select 4 random indexes that within the RANK_RETENTION_RATE
        int c1 = (hiprand(&state) % (int)(numChromosomes * RANK_RETENTION_RATE));
        int c2 = (hiprand(&state) % (int)(numChromosomes * RANK_RETENTION_RATE));
        int c3 = (hiprand(&state) % (int)(numChromosomes * RANK_RETENTION_RATE));
        int c4 = (hiprand(&state) % (int)(numChromosomes * RANK_RETENTION_RATE));

        // best ranks between c1/c2 and c3/c4 will swap
        dev_swap_candidates[(blockIdx.x * 2)] = c1 < c2 ? c1 : c2;
        dev_swap_candidates[(blockIdx.x * 2) + 1] = c3 < c4 ? c3 : c4;

        // decide which subgrid we will start the swap at
        // this number is in the range [2,dimension-1], don't start at first or last subgrid
        dev_swap_index[blockIdx.x] = (blockIdx.x % (dimension - 2)) + 1;
    }

    __syncthreads();

    int offset = ((blockIdx.x) * numGenes) + threadIdx.x; // location of gene in flattened population
    int grid = GET_SUB_GRID(threadIdx.x, subDim); // determines the grid of current thread within a block
    
    // Get new offset by picking the correct swap candidate
    // if the current grid is less than the swap index, its the first candidate, otherwise its the second
    int offset_new = (dev_swap_candidates[(blockIdx.x * 2) + (grid < dev_swap_index[blockIdx.x] ? 0 : 1)] * numGenes) + threadIdx.x;
    flattenedPop[offset] = dev_tempPopualtion[offset_new];
    
    __syncthreads();

    // This carries out a randome number of 'mutations' 
    // a mutation is just swapping to non-locked genes of a subgrid
    if (threadIdx.x == 0)
    {
        // 1 to 3 mutations will happen on every block (this was randomly decided)
        // I was messing with this a lot and this seems to be best so far
        for (int k = 0; k < (hiprand(&state) % subDim) + 1; k++)
        {
            // randomly generates a swap index and makes sure that it is not locked gene
            int swap_index_1 = (hiprand(&state) % numGenes);
            while (lockedIn[swap_index_1]) swap_index_1 = (hiprand(&state) % numGenes);

            // determines the subgrid number of swap_index_1, then determines the center of that subgrid
            int swap_grid = GET_SUB_GRID(swap_index_1, subDim);

            int rowOffset = subDim * (swap_grid / subDim); // Subgrid tiles logic
            int colOffset = subDim * (swap_grid % subDim);

            int swap_index_2 = swap_index_1;
            int subGridTile = 0;

            do
            {
                // Randomly choose unlocked tile within subgrid
                // Thought of using incremental logic here to guarantee it completes in finite time but that makes certain tiles more probable for swaps depending on locked genes
                subGridTile = hiprand(&state) % dimension;
                swap_index_2 = (rowOffset + (subGridTile / subDim)) * dimension + colOffset + (subGridTile % subDim);
            } while (swap_index_1 == swap_index_2 || lockedIn[swap_index_2]);

            // swaps swap_index_1 and swap_index_2
            char temp = flattenedPop[(blockIdx.x * numGenes) + swap_index_1];
            flattenedPop[(blockIdx.x * numGenes) + swap_index_1] = flattenedPop[(blockIdx.x * numGenes) + swap_index_2];
            flattenedPop[(blockIdx.x * numGenes) + swap_index_2] = temp;
        }
    }
}

Population* Breed(Population *popIn, int &bestrank, char* bestboard)
{
    int numChromosomes = 0;
    int numGenes = 0;
    std::shared_ptr<bool[]> lockedGenesIn = popIn->GetLockedGenes();

    char *flattenedPop = popIn->FlattenPopulationToArray(numChromosomes, numGenes, false);

    int dimension = sqrt(numGenes);
    int subDim = sqrt(dimension);

    // START FITNESS RANK
    int *errorCounts = new int[numChromosomes];

    char *dev_flattenedPop;
    int *dev_fitnessCount;

    hipMalloc((void **)&dev_flattenedPop, numChromosomes * numGenes * sizeof(char));
    hipMalloc((void **)&dev_fitnessCount, numChromosomes * sizeof(int));

    hipMemcpy(dev_flattenedPop, flattenedPop, numChromosomes * numGenes * sizeof(char), hipMemcpyHostToDevice);

    auto startFitness = std::chrono::high_resolution_clock::now();
    RankFitnessKernel<<<numChromosomes, numGenes >>>(numChromosomes, dimension, dev_flattenedPop, dev_fitnessCount);
    auto stopFitness = std::chrono::high_resolution_clock::now();

    hipMemcpy(errorCounts, dev_fitnessCount, numChromosomes * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_fitnessCount);

    // Give chromosomes a unique rank based on their errorCount/fitnessScore

    // Using std::sort is O(n log n), previous algorithm was O(n^2)
    // <fitnessScore, chromosomeIndex>
    std::vector<std::pair<int, int>> scoreToIndex = std::vector<std::pair<int, int>>();

    for (int i = 0; i < numChromosomes; ++i)
    {
        scoreToIndex.push_back(std::make_pair(errorCounts[i], i));
    }

    // Sorts by fitnessScore, O(log n)
    std::sort(scoreToIndex.begin(), scoreToIndex.end());

    int *fitnessRanks = new int[numChromosomes];

    int i = 0;
    for (auto &pair : scoreToIndex)
    {
        fitnessRanks[pair.second] = i++;
    }

    // END FITNESS RANK

    // This is just used for printing the best solution at the end    
    int prev_best = bestrank;
    bestrank = INT_MAX;
    int index = 0;
    for (int e = 0; e < numChromosomes; e++)
    {
        if (errorCounts[e] < bestrank) 
        {
            bestrank = errorCounts[e];
            index = e;
            if (bestrank < prev_best)
            {
                for (int i = 0; i < dimension; i++)
                {
                    for (int j = 0; j < dimension; j++)
                    {
                        bestboard[(i * dimension) + j] = flattenedPop[(index * numGenes) + (i * dimension) + j];
                    }
                }
            }
        }
        
    }

    std::cout << "Best error - " << bestrank << "\n";
    
    char *dev_tempPopualtion;
    int *dev_ranks;
    bool *dev_lockedIn;
    int *dev_swap_index;
    int *dev_swap_candidates;

    int retention_size = (int)(numChromosomes * RANK_RETENTION_RATE);

    hipMalloc((void **)&dev_tempPopualtion, retention_size * numGenes * sizeof(char));
    hipMalloc((void **)&dev_ranks, numChromosomes * sizeof(int));
    hipMalloc((void **)&dev_lockedIn, numChromosomes * sizeof(bool));
    hipMalloc((void **)&dev_swap_index, numChromosomes * sizeof(int));
    hipMalloc((void **)&dev_swap_candidates, numChromosomes * 2 * sizeof(int));

    hipMemcpy(dev_ranks, fitnessRanks, numChromosomes * sizeof(int), hipMemcpyHostToDevice);
    
    hipMemcpy(dev_lockedIn, lockedGenesIn.get(), numChromosomes * sizeof(bool), hipMemcpyHostToDevice);

    auto startBreed = std::chrono::high_resolution_clock::now();
    BreedKernel<<<numChromosomes, numGenes>>>(numChromosomes, numGenes, dimension, subDim, (rand() % dimension) + 1, dev_flattenedPop, dev_ranks, dev_lockedIn, dev_swap_index, dev_swap_candidates, dev_tempPopualtion);
    auto stopBreed = std::chrono::high_resolution_clock::now();

    std::cout << "Parallel Time: " << ((std::chrono::duration_cast<std::chrono::microseconds>(stopFitness - startFitness)).count() + (std::chrono::duration_cast<std::chrono::microseconds>(stopBreed - startBreed)).count()) << "\n";

    char *popout = new char[numChromosomes * numGenes];
    hipMemcpy(popout, dev_flattenedPop, numChromosomes * numGenes * sizeof(char), hipMemcpyDeviceToHost);
    
    hipFree(dev_flattenedPop);
    hipFree(dev_ranks);
    hipFree(dev_lockedIn);
    hipFree(dev_tempPopualtion);
    hipFree(dev_swap_index);
    hipFree(dev_swap_candidates);

    delete[] errorCounts;
    delete[] fitnessRanks;

    Population *out = new Population(numGenes, numChromosomes, lockedGenesIn, popout);

    return out;
}