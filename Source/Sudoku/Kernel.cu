#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "Sudoku/Kernel.cuh"

__global__ void PredetermineTilesKernel(int subDim, int dimension, char *boardIn, char *boardOut)
{
    int tileId = threadIdx.x;
    int row = tileId / dimension;
    int col = tileId % dimension;
    int subGrid = GET_SUB_GRID(tileId, subDim);
    int rowOffset = subDim * (subGrid / subDim); // Subgrid tiles logic
    int colOffset = subDim * (subGrid % subDim);

    __shared__ bool modified[MAX_DIM * MAX_DIM];
    __shared__ bool madeChange;

    do
    {
        modified[threadIdx.x] = false;

        if (boardIn[tileId] != 0)
        {
            // Tile already has a set value, skip it
            boardOut[tileId] = boardIn[tileId];
        }
        else
        {
            bool usedNums[MAX_DIM];
            for (int iVal = 0; iVal < dimension; ++iVal)
            {
                // Every tile in column
                char tile = boardIn[iVal * dimension + col];
                if (tile > 0)
                {
                    usedNums[tile - 1] = true;
                }

                // Every tile in row
                tile = boardIn[row * dimension + iVal];
                if (tile > 0)
                {
                    usedNums[tile - 1] = true;
                }

                // Every tile in subgrid, convert iVal into 1D index of board
                tile = boardIn[(rowOffset + (iVal / subDim)) * dimension + colOffset + (iVal % subDim)];
                if (tile > 0)
                {
                    usedNums[tile - 1] = true;
                }
            }

            char candidate = 0;
            for (int i = 0; i < dimension; i++)
            {
                if (!usedNums[i])
                {
                    if (candidate != 0)
                    {
                        // More than one candidate value
                        candidate = 0;
                        break;
                    }
                    else
                    {
                        candidate = i + 1;
                    }
                }
            }

            boardOut[tileId] = candidate;
            modified[threadIdx.x] = candidate != 0;
        }

        __syncthreads();

        if (threadIdx.x == 0)
        {
            madeChange = false;
            for (int i = 0; i < dimension * dimension; ++i)
            {
                if (modified[i])
                {
                    madeChange = true;
                    break;
                }
            }
        }

        __syncthreads();

        // Running another iteration, copy current output for next input
        if (madeChange)
        {
            boardIn[tileId] = boardOut[tileId];
        }

        __syncthreads();

    } while (madeChange);
}

Board* PredetermineTiles(Board *boardIn)
{
    int dimension = boardIn->GetDimension();
    int subDim = sqrt(dimension);
    char *boardArrIn = boardIn->GetBoardPointer();

    char *dev_boardIn, *dev_boardOut;

    hipMalloc((void **)&dev_boardIn, dimension * dimension * sizeof(char));
    hipMalloc((void **)&dev_boardOut, dimension * dimension * sizeof(char));

    hipMemcpy(dev_boardIn, boardArrIn, dimension * dimension * sizeof(char), hipMemcpyHostToDevice);
    
    PredetermineTilesKernel<<<1, dimension * dimension>>>(subDim, dimension, dev_boardIn, dev_boardOut);

    char *boardArrOut = new char[dimension * dimension];
    hipMemcpy(boardArrOut, dev_boardOut, dimension * dimension * sizeof(char), hipMemcpyDeviceToHost);

    hipFree(dev_boardIn);
    hipFree(dev_boardOut);

    Board *out = new Board(dimension, boardArrOut);
    return out;
}



__global__ void RankFitnessKernel(int dimension, char *flattenedPop, int *fitnessRankOut)
{
    int tileId = threadIdx.x;
    int chromOffset = blockIdx.x * blockDim.x;
    int row = tileId / dimension;
    int col = tileId % dimension;

    __shared__ int errors[MAX_DIM * MAX_DIM];
    errors[threadIdx.x] = 0;

    char currTile = flattenedPop[chromOffset + tileId];

    // Count how many tiles in col/row are same (conflicting) with current tile
    for (int iVal = 0; iVal < dimension; ++iVal)
    {
        int colTile = iVal * dimension + col;
        int rowTile = row * dimension + iVal;

        if (colTile != tileId && flattenedPop[chromOffset + colTile] == currTile)
        {
            ++errors[threadIdx.x];
        }

        if (rowTile != tileId && flattenedPop[chromOffset + rowTile] == currTile)
        {
            ++errors[threadIdx.x];
        }
    }

    __syncthreads();

    // Parallel reduction
    if (threadIdx.x % dimension == 0)
    {
        for (int i = 1; i < dimension; ++i)
        {
            errors[threadIdx.x] += errors[threadIdx.x + i];
        }
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        fitnessRankOut[blockIdx.x] = 0;
        for (int i = 0; i < dimension; ++i)
        {
            fitnessRankOut[blockIdx.x] += errors[i * dimension];
        }
    }
}

int* RankFitness(Population *popIn)
{
    int numChromosomes = 0;
    int numGenes = 0;

    // Arguments are output args, filled by function
    char *flattenedPop = popIn->FlattenPopulationToArray(numChromosomes, numGenes, false);

    int dimension = sqrt(numGenes);

    char *dev_flattenedPop;
    int *dev_fitnessRank;

    hipMalloc((void **)&dev_flattenedPop, numChromosomes * numGenes * sizeof(char));
    hipMalloc((void **)&dev_fitnessRank, numChromosomes * sizeof(int));

    hipMemcpy(dev_flattenedPop, flattenedPop, numChromosomes * numGenes * sizeof(char), hipMemcpyHostToDevice);

    RankFitnessKernel<<<numChromosomes, numGenes>>>(dimension, dev_flattenedPop, dev_fitnessRank);

    int *fitnessRank = new int[numChromosomes];
    hipMemcpy(fitnessRank, dev_fitnessRank, numChromosomes * sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(dev_flattenedPop);
    hipFree(dev_fitnessRank);

    return fitnessRank;
}
