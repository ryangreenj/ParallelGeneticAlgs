#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "Sudoku/Kernel.cuh"
#include <iostream>
#include <algorithm>    // std::shuffle
#include <array>        // std::array
#include <random>       // std::default_random_engine
#include <chrono>       // std::chrono::system_clock
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void PredetermineTilesKernel(int subDim, int dimension, char *boardIn, char *boardOut)
{
    int tileId = threadIdx.x;
    int row = tileId / dimension;
    int col = tileId % dimension;
    int subGrid = GET_SUB_GRID(tileId, subDim);
    int rowOffset = subDim * (subGrid / subDim); // Subgrid tiles logic
    int colOffset = subDim * (subGrid % subDim);

    __shared__ bool modified[MAX_DIM * MAX_DIM];
    __shared__ bool madeChange;

    do
    {
        modified[threadIdx.x] = false;

        if (boardIn[tileId] != 0)
        {
            // Tile already has a set value, skip it
            boardOut[tileId] = boardIn[tileId];
        }
        else
        {
            bool usedNums[MAX_DIM];
            for (int iVal = 0; iVal < dimension; ++iVal)
            {
                // Every tile in column
                char tile = boardIn[iVal * dimension + col];
                if (tile > 0)
                {
                    usedNums[tile - 1] = true;
                }

                // Every tile in row
                tile = boardIn[row * dimension + iVal];
                if (tile > 0)
                {
                    usedNums[tile - 1] = true;
                }

                // Every tile in subgrid, convert iVal into 1D index of board
                tile = boardIn[(rowOffset + (iVal / subDim)) * dimension + colOffset + (iVal % subDim)];
                if (tile > 0)
                {
                    usedNums[tile - 1] = true;
                }
            }

            char candidate = 0;
            for (int i = 0; i < dimension; i++)
            {
                if (!usedNums[i])
                {
                    if (candidate != 0)
                    {
                        // More than one candidate value
                        candidate = 0;
                        break;
                    }
                    else
                    {
                        candidate = i + 1;
                    }
                }
            }

            boardOut[tileId] = candidate;
            modified[threadIdx.x] = candidate != 0;
        }

        __syncthreads();

        if (threadIdx.x == 0)
        {
            madeChange = false;
            for (int i = 0; i < dimension * dimension; ++i)
            {
                if (modified[i])
                {
                    madeChange = true;
                    break;
                }
            }
        }

        __syncthreads();

        // Running another iteration, copy current output for next input
        if (madeChange)
        {
            boardIn[tileId] = boardOut[tileId];
        }

        __syncthreads();

    } while (madeChange);
}

Board* PredetermineTiles(Board *boardIn)
{
    int dimension = boardIn->GetDimension();
    int subDim = sqrt(dimension);
    char *boardArrIn = boardIn->GetBoardPointer();

    char *dev_boardIn, *dev_boardOut;

    hipMalloc((void **)&dev_boardIn, dimension * dimension * sizeof(char));
    hipMalloc((void **)&dev_boardOut, dimension * dimension * sizeof(char));

    hipMemcpy(dev_boardIn, boardArrIn, dimension * dimension * sizeof(char), hipMemcpyHostToDevice);
    
    PredetermineTilesKernel<<<1, dimension * dimension>>>(subDim, dimension, dev_boardIn, dev_boardOut);

    char *boardArrOut = new char[dimension * dimension];
    hipMemcpy(boardArrOut, dev_boardOut, dimension * dimension * sizeof(char), hipMemcpyDeviceToHost);

    hipFree(dev_boardIn);
    hipFree(dev_boardOut);

    Board *out = new Board(dimension, boardArrOut);
    return out;
}



__global__ void RankFitnessKernel(int chromosomes, int dimension, char *flattenedPop, int *fitnessCount)
{
    int tileId = threadIdx.x;
    int chromOffset = blockIdx.x * blockDim.x;
    int row = tileId / dimension;
    int col = tileId % dimension;

    __shared__ int errors[MAX_DIM * MAX_DIM];
    errors[threadIdx.x] = 0;

    char currTile = flattenedPop[chromOffset + tileId];

    // Count how many tiles in col/row are same (conflicting) with current tile
    for (int iVal = 0; iVal < dimension; ++iVal)
    {
        int colTile = iVal * dimension + col;
        int rowTile = row * dimension + iVal;

        if (colTile != tileId && flattenedPop[chromOffset + colTile] == currTile)
        {
            ++errors[threadIdx.x];
        }

        if (rowTile != tileId && flattenedPop[chromOffset + rowTile] == currTile)
        {
            ++errors[threadIdx.x];
        }
    }

    __syncthreads();

    // Parallel reduction
    if (threadIdx.x % dimension == 0)
    {
        for (int i = 1; i < dimension; ++i)
        {
            errors[threadIdx.x] += errors[threadIdx.x + i];
        }
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        fitnessCount[blockIdx.x] = 0;
        for (int i = 0; i < dimension; ++i)
        {
            fitnessCount[blockIdx.x] += errors[i * dimension];
        }
    }

}

int* RankFitness(int numChromosomes, int numGenes, char *flattenedPop, int *errorCountsOut)
{
    // int numChromosomes = 0;
    // int numGenes = 0;

    // Arguments are output args, filled by function
    // char *flattenedPop = popIn->FlattenPopulationToArray(numChromosomes, numGenes, false);

    int dimension = sqrt(numGenes);

    char *dev_flattenedPop;
    int *dev_fitnessCount;

    hipMalloc((void **)&dev_flattenedPop, numChromosomes * numGenes * sizeof(char));
    hipMalloc((void **)&dev_fitnessCount, numChromosomes * sizeof(int));

    hipMemcpy(dev_flattenedPop, flattenedPop, numChromosomes * numGenes * sizeof(char), hipMemcpyHostToDevice);

    RankFitnessKernel<<<numChromosomes, numGenes>>>(numChromosomes, dimension, dev_flattenedPop, dev_fitnessCount);

    int *fitnessRank = new int[numChromosomes];

    hipMemcpy(errorCountsOut, dev_fitnessCount, numChromosomes * sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(dev_flattenedPop);
    hipFree(dev_fitnessCount);

    // This can be parallelized but I was having some mem issues
    for (int i = 0; i < numChromosomes; ++i)
    {
        fitnessRank[i] = 0;
        for (int j = 0; j < numChromosomes; ++j)
        {   
            if (errorCountsOut[i] > errorCountsOut[j])
            {
                fitnessRank[i] += 1;
            }
        }
    }

    int* rank_set = new int[numChromosomes];
    // for (int i = 0; i < numChromosomes; ++i)
    // {
    //     rank_set[i] = -999;
    // }
    
    for (int i = 0; i < numChromosomes; ++i)
    {
        bool inSet = false;
        for (int j = 0; j < i; ++j)
        {
            if (rank_set[j] == fitnessRank[i])
            {
                inSet = true;
                break;
            }            
        }

        if (inSet)
        {
            fitnessRank[i] += 1;
        } 
        rank_set[i] = fitnessRank[i];
    }
    return fitnessRank;
}

__global__ void BreedKernel(int numChromosomes, int numGenes, int seed, char *flattenedPop, int *subgrid_swaps, bool *lockedIn, char *flattenedPopOut)
{
    int grid = ((threadIdx.x % 9) / 3) + ((threadIdx.x / 27) * 3); // Determines the grid of current run
    int offset = ((blockIdx.x) * numGenes) + threadIdx.x;
    int offset_new = ((subgrid_swaps[blockIdx.x]) * numGenes) + threadIdx.x;

    // Determines random subgrid to swap if subgrid swap id is not the block id
    int grid_swap = (blockIdx.x == subgrid_swaps[blockIdx.x]) ? blockIdx.x : (blockIdx.x + subgrid_swaps[blockIdx.x] * seed) % 8;
    flattenedPopOut[offset] = flattenedPop[(grid <= grid_swap) ? offset_new : offset ];

    __syncthreads();

    if (threadIdx.x == 0 && subgrid_swaps[blockIdx.x] != blockIdx.x)
    {   
        // Random mutations
        hiprandState_t state;

        /* we have to initialize the state */
        hiprand_init(0, /* the seed controls the sequence of random values that are produced */
                    blockIdx.x, /* the sequence number is only important with multiple cores */
                    0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                    &state);

        for (int k = 0; k < 3; k++)
        {
            int swap_index_1 = (hiprand(&state) % 81);
         
            while(lockedIn[swap_index_1]) 
            {
                swap_index_1 = (hiprand(&state) % 81);
            }

            int swap_grid = ((swap_index_1 % 9) / 3) + ((swap_index_1 / 27) * 3); 
            int swap_grid_center = 10 + ((swap_grid / 3) * 27) + ((swap_grid % 3) * 3);

            int swap_index_2 = swap_index_1;

            do
            {
                int y_shift = (hiprand(&state) % 3) - 1;
                int x_shift = (hiprand(&state) % 3) - 1;

                swap_index_2 = swap_grid_center + (9 * y_shift) + x_shift;
            } while(swap_index_1 == swap_index_2 || lockedIn[swap_index_2]);
            
            char temp = flattenedPopOut[(blockIdx.x * 81) + swap_index_1];

            flattenedPopOut[(blockIdx.x * 81) + swap_index_1] = flattenedPopOut[(blockIdx.x * 81) + swap_index_2];
            flattenedPopOut[(blockIdx.x * 81) + swap_index_2] = temp;
            // flattenedPopOut[(blockIdx.x * 81) + swap_index_1] = 99;
            // flattenedPopOut[(blockIdx.x * 81) + swap_index_2] = 99;
        }
    }
}

Population* Breed(Population *popIn)
{
    double choices[2] = {0.20, 0.8}; // Keep top 20%, breed next 60%, drop final 20%

    int numChromosomes = 0;
    int numGenes = 0;
    std::shared_ptr<bool[]> lockedGenesIn = popIn->GetLockedGenes();

    // Arguments are output args, filled by function
    // Need to delete returned pointer at end
    char *flattenedPop = popIn->FlattenPopulationToArrayShuffle(numChromosomes, numGenes);

    int *errorCounts = new int[numChromosomes];
    int* fitnessRanks = RankFitness(numChromosomes, numGenes, flattenedPop, errorCounts);

    int min_ = 999;
    for (int e = 0; e < numChromosomes; e++)
    {
        if (errorCounts[e] < min_) min_ = errorCounts[e];
    }

    std::cout << "Best error - " << min_ << "\n";
    // std::cout << "r ";
    // for (int i = 0; i < numChromosomes; i++) std::cout << i << " " << fitnessRanks[i] << ", ";
    // std::cout << "\n";


    char *dev_flattenedPopBreed;
    char *dev_flattenedPopBreedOut;
    int *dev_swaps;
    bool *dev_lockedIn;
    int *grid_swap = new int[numChromosomes];

    
    
    int* swaps = new int[numChromosomes];
    bool* stay = new bool[numChromosomes];

    for (int i = 0; i < numChromosomes; i++) stay[i] = fitnessRanks[i] < (int)(.2 * numChromosomes);

    for (int i = 0; i < numChromosomes; i++){
        if(fitnessRanks[i] < (int)(.2 * numChromosomes))
        {
            swaps[i] = i;
        }
        else 
        {
            int i1 = i;
            do
            {
                i++;
                if (i >= numChromosomes) break;
                swaps[i] = i;
            } while (fitnessRanks[i] < (int)(.2 * numChromosomes));
            
            int i2 = i;

            swaps[i1] = i2;
            swaps[i2] = i1;
        }
    }

    // std::cout << "s ";
    // for (int i = 0; i < numChromosomes; i++) std::cout << i << " " << swaps[i] << ", ";
    // std::cout << "\n";

    hipMalloc((void **)&dev_flattenedPopBreed, numChromosomes * numGenes * sizeof(char));
    hipMalloc((void **)&dev_flattenedPopBreedOut, numChromosomes * numGenes * sizeof(char));
    hipMalloc((void **)&dev_swaps, numChromosomes * sizeof(int));
    hipMalloc((void **)&dev_lockedIn, numChromosomes * sizeof(bool));

    hipMemcpy(dev_flattenedPopBreed, flattenedPop, numChromosomes * numGenes * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_swaps, swaps, numChromosomes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_lockedIn, lockedGenesIn.get(), numChromosomes * sizeof(bool), hipMemcpyHostToDevice);

    BreedKernel<<<numChromosomes, numGenes>>>(numChromosomes, numGenes, (rand() % 9) + 1, dev_flattenedPopBreed, dev_swaps, dev_lockedIn, dev_flattenedPopBreedOut);
    
    char *popout = new char[numChromosomes * numGenes];
    hipMemcpy(popout, dev_flattenedPopBreedOut, numChromosomes * numGenes * sizeof(char), hipMemcpyDeviceToHost);
    
    // hipFree(dev_flattenedPopBreed);
    // hipFree(dev_flattenedPopBreedOut);
    // hipFree(dev_swaps);
    // hipFree(dev_lockedIn);

    Population *out = new Population(numGenes, numChromosomes, lockedGenesIn, popout);
    return out;
}